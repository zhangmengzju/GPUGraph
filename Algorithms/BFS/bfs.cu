#include "hip/hip_runtime.h"
/**
 Copyright 2013-2014 SYSTAP, LLC.  http://www.systap.com

 Licensed under the Apache License, Version 2.0 (the "License");
 you may not use this file except in compliance with the License.
 You may obtain a copy of the License at

 http://www.apache.org/licenses/LICENSE-2.0

 Unless required by applicable law or agreed to in writing, software
 distributed under the License is distributed on an "AS IS" BASIS,
 WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 See the License for the specific language governing permissions and
 limitations under the License.

 This work was (partially) funded by the DARPA XDATA program under
 AFRL Contract #FA8750-13-C-0002.

 This material is based upon work supported by the Defense Advanced
 Research Projects Agency (DARPA) under Contract No. D14PC00029.
 */

typedef unsigned int uint;
#include <stdio.h> 
#include <cstdlib>
#include <algorithm>
#include <iostream>
#include <fstream>
#include <string>
#include <deque>
#include <vector>
#include <bfs.h>
#include <iostream>
#include <time.h>

#include <config.h>

// Utilities and correctness-checking
//#include <test/b40c_test_util.h>

// Graph construction utils

#include <b40c/graph/builder/market.cuh>
#include <b40c/graph/builder/random.cuh>
#include <b40c/graph/builder/rmat.cuh>

#include <GASengine/csr_problem.cuh>
#include <GASengine/enactor_vertex_centric.cuh>
#include <predextract.cuh>

using namespace b40c;
using namespace graph;
using namespace std;

template<typename VertexId, typename Value, typename SizeT>
void CPUBFS(int directed, int test_iteration,
		const CsrGraph<VertexId, Value, SizeT> &csr_graph,
		VertexId *source_path, VertexId src) {
	printf("Running CPU code...");
	fflush (stdout);

	// (Re)initialize distances
	for (VertexId i = 0; i < csr_graph.nodes; i++) {
		source_path[i] = -1;
	}
	source_path[src] = 0;
	VertexId search_depth = 0;

	const time_t startTime = time(NULL);

	// Initialize queue for managing previously-discovered nodes
	std::deque<VertexId> frontier;
	frontier.push_back(src);

	//
	// Perform BFS on CPU
	//

	int iter = 0;
	while (!frontier.empty()) {
		// Dequeue node from frontier
		VertexId dequeued_node = frontier.front();
		frontier.pop_front();
		VertexId neighbor_dist = source_path[dequeued_node] + 1;

		// Locate adjacency list
		int edges_begin = csr_graph.row_offsets[dequeued_node];
		int edges_end = csr_graph.row_offsets[dequeued_node + 1];
		for (int edge = edges_begin; edge < edges_end; edge++) {

			// Lookup neighbor and enqueue if undiscovered
			VertexId neighbor = csr_graph.column_indices[edge];
			if (source_path[neighbor] == -1) {
				source_path[neighbor] = neighbor_dist;
				if (search_depth < neighbor_dist) {
					search_depth = neighbor_dist;
				}
				frontier.push_back(neighbor);
			}
		}

		if (directed == 0) {
			edges_begin = csr_graph.column_offsets[dequeued_node];
			edges_end = csr_graph.column_offsets[dequeued_node + 1];
			for (int edge = edges_begin; edge < edges_end; edge++) {

				// Lookup neighbor and enqueue if undiscovered
				VertexId neighbor = csr_graph.row_indices[edge];
				if (source_path[neighbor] == -1) {
					source_path[neighbor] = neighbor_dist;
					if (search_depth < neighbor_dist) {
						search_depth = neighbor_dist;
					}
					frontier.push_back(neighbor);
				}
			}
		}
		iter++;
	}

	const time_t EndTime = time(NULL);

	std::cout << "CPU time took: " << difftime(EndTime, startTime) * 1000
			<< " ms" << std::endl;
	search_depth++;
}

bool cudaInit(int device) {
	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess) {
		printf("hipGetDeviceCount returned %d\n-> %s\n", (int) error_id,
				hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit (EXIT_FAILURE);
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0) {
		printf("There are no available device(s) that support CUDA\n");
		return false;
	} else {
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}

	int dev, driverVersion = 0, runtimeVersion = 0;

	for (dev = 0; dev < deviceCount; ++dev) {
		if (dev == device) {
			hipSetDevice(dev);
			hipDeviceProp_t deviceProp;
			hipGetDeviceProperties(&deviceProp, dev);

			printf("Running on this device:");
			printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

			// Console log
			hipDriverGetVersion(&driverVersion);
			hipRuntimeGetVersion(&runtimeVersion);
			printf(
					"  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
					driverVersion / 1000, (driverVersion % 100) / 10,
					runtimeVersion / 1000, (runtimeVersion % 100) / 10);
			printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
					deviceProp.major, deviceProp.minor);

			printf(
					"  Total amount of global memory:                 %.0f MBytes (%llu bytes)\n",
					(float) deviceProp.totalGlobalMem / 1048576.0f,
					(unsigned long long) deviceProp.totalGlobalMem);

			break;
		}
	}

	return true;
}

bool correctTest(int nodes, int* reference_labels, int* h_labels) {
	bool pass = true;
	printf("Correctness testing ...");
	fflush (stdout);
	for (int i = 0; i < nodes; i++) {
		if (reference_labels[i] != h_labels[i]) {
			//      printf("Incorrect value for node %d: CPU value %d, GPU value %d\n", i, reference_labels[i], h_labels[i]);
			pass = false;
		}
	}
	if (pass)
		printf("passed\n");
	else
		printf("failed\n");
	return pass;
}

void printUsageAndExit(char *algo_name) {
	std::cout << "Usage: " << algo_name
			<< " [-graph (-g) graph_file] [-output (-o) output_file] [-sources src_file] [-BFS \"variable1=value1 variable2=value2 ... variable3=value3\" -help ] [-c config_file]\n";
	std::cout << "     -help display the command options\n";
	std::cout
			<< "     -graph specify a sparse matrix in Matrix Market (.mtx) format\n";
	std::cout << "     -output or -o specify file for output result\n";
	std::cout << "     -sources or -s set starting vertices file\n";
	std::cout << "     -c set the BFS options from the configuration file\n";
	std::cout
			<< "     -parameters (-p) set the options.  Options include the following:\n";
	Config::printOptions();

	exit(0);
}

int main(int argc, char **argv) {

	const char* outFileName = 0;
	//  int src[1];
	//  bool g_undirected;
	const bool g_stream_from_host = false;
	const bool g_with_value = true;
	const bool g_mark_predecessor = false;
	bool g_verbose = false;
	typedef bfs::VertexId VertexId; // Use as the node identifier type
	typedef bfs::DataType Value; // Use as the value type
	typedef bfs::SizeT SizeT; // Use as the graph size type
	char* graph_file = NULL;
	CsrGraph<VertexId, Value, SizeT> csr_graph(g_stream_from_host);
	char source_file_name[100] = "";
	//  int device = 0;
	//  double max_queue_sizing = 1.3;
	Config cfg;

	for (int i = 1; i < argc; i++) {
		if (strncmp(argv[i], "-help", 100) == 0) // print the usage information
			printUsageAndExit(argv[0]);
		else if (strncmp(argv[i], "-graph", 100) == 0
				|| strncmp(argv[i], "-g", 100) == 0) { //input graph
			i++;

			graph_file = argv[i];

		} else if (strncmp(argv[i], "-output", 100) == 0
				|| strncmp(argv[i], "-o", 100) == 0) { //output file name
			i++;
			outFileName = argv[i];
		}

		else if (strncmp(argv[i], "-sources", 100) == 0
				|| strncmp(argv[i], "-s", 100) == 0) { //the file containing starting vertices
			i++;
			strcpy(source_file_name, argv[i]);
		}

		else if (strncmp(argv[i], "-parameters", 100) == 0
				|| strncmp(argv[i], "-p", 100) == 0) { //The BFS specific options
			i++;
			cfg.parseParameterString(argv[i]);
		} else if (strncmp(argv[i], "-c", 100) == 0) { //use a configuration file to specify the BFS options instead of command line
			i++;
			cfg.parseFile(argv[i]);
		}
	}

	if (graph_file == NULL) {
		printUsageAndExit(argv[0]);
		exit(1);
	}

	int device_id = cfg.getParameter<int>("device");
	bool cudaEnabled = cudaInit(device_id);
	if (!cudaEnabled)
		return 0;

	char hostname[1024] = "localhost";
#ifdef gethostname
	gethostname(hostname, 1023);
#endif

	printf("Running on host: %s\n", hostname);

	int directed = cfg.getParameter<int>("directed");

	/*
	 if(strcmp(graph_file, "RMAT") == 0)
	 {
	 int tmpn = pow(2,21);
	 int tmpe = tmpn * 16;
	 if(builder::BuildRmatGraph<g_with_value>(tmpn, tmpe, csr_graph, true, 0.45, 0.15, 0.15) != 0)
	 exit(1);
	 }
	 else*/
	{
		if (builder::BuildMarketGraph<g_with_value>(graph_file, csr_graph,
				false) != 0)
			exit(1);
	}
	//  csr_graph.DisplayGraph();
	int num_srcs = 0;
	int* srcs = NULL;
	int origin = cfg.getParameter<int>("origin");
	int iter_num = cfg.getParameter<int>("iter_num");
	int threshold = cfg.getParameter<int>("threshold");

	const int max_src_num = 1000;

	if (strcmp(source_file_name, "")) {
		if (strcmp(source_file_name, "RANDOM") == 0) {
			printf("Using random starting vertices!\n");
			num_srcs = cfg.getParameter<int>("num_src");
			srcs = new int[num_srcs];
			printf("Using %d random starting vertices!\n", num_srcs);
			srand (time(NULL));int
			count = 0;
			while (count < num_srcs) {
				int tmp_src = rand() % csr_graph.nodes;
				if (csr_graph.row_offsets[tmp_src + 1]
						- csr_graph.row_offsets[tmp_src] > 0) {
					srcs[count++] = tmp_src;
				}
			}

		} else {
			printf("Using source file: %s!\n", source_file_name);
			FILE* src_file;
			if ((src_file = fopen(source_file_name, "r")) == NULL) {
				printf("Source file open error!\n");
				exit(0);
			}

			srcs = new int[max_src_num];
			for (num_srcs = 0; num_srcs < max_src_num; num_srcs++) {
				if (fscanf(src_file, "%d\n", &srcs[num_srcs]) != EOF) {
					if (origin == 1)
						srcs[num_srcs]--; //0-based index
				} else
					break;
			}
			printf("number of srcs used: %d\n", num_srcs);
		}

	} else {
		int src_node = cfg.getParameter<int>("src");
		int origin = cfg.getParameter<int>("origin");
		num_srcs = 1;
		srcs = new int[1];
		srcs[0] = src_node;
		if (origin == 1)
			srcs[0]--;
		printf("Single source vertex: %d\n", srcs[0]);
	}

	{
		const int stats = cfg.getParameter<int>("stats");
		if (stats) {
			csr_graph.PrintHistogram();
		}
	}

	VertexId* reference_labels;

	int run_CPU = cfg.getParameter<int>("run_CPU");
	if (strcmp(source_file_name, "") == 0 && run_CPU) //Do correctness test only with single starting vertex
			{
		reference_labels = (VertexId*) malloc(
				sizeof(VertexId) * csr_graph.nodes);
		int test_iteration = 1;
		int src = cfg.getParameter<int>("src");

		if (origin == 1)
			src--;

		CPUBFS(directed, test_iteration, csr_graph, reference_labels, src);
	}

	// Allocate problem on GPU
	int num_gpus = 1;
	typedef GASengine::CsrProblem<bfs, VertexId, SizeT, Value,
			g_mark_predecessor, g_with_value> CsrProblem;
	CsrProblem csr_problem(cfg);

	if (csr_problem.FromHostProblem(g_stream_from_host, csr_graph.nodes,
			csr_graph.edges, csr_graph.column_indices, csr_graph.row_offsets,
			csr_graph.edge_values, csr_graph.row_indices,
			csr_graph.column_offsets, num_gpus, directed))
		exit(1);

	const bool INSTRUMENT = true;

	GASengine::EnactorVertexCentric<CsrProblem, bfs, INSTRUMENT> vertex_centric(
			cfg, g_verbose);

	for (int i = 0; i < num_srcs; i++) {
		int tmpsrcs[1];
		tmpsrcs[0] = srcs[i];
		printf("num_srcs=%d, src=%d, iter_num=%d\n", num_srcs, tmpsrcs[i],
				iter_num);

		hipError_t retval = hipSuccess;

		retval = vertex_centric.EnactIterativeSearch(csr_problem,
				csr_graph.row_offsets, directed, 1, tmpsrcs, iter_num,
				threshold);

		if (retval && (retval != hipErrorInvalidDeviceFunction)) {
			exit(1);
		}
	}

	int* d_predlist;
	hipMalloc((void**) &d_predlist, csr_graph.nodes * sizeof(int));

	//extract the predecessor list: the predecessor computed for the src vertex is arbitrary and should be ignored.
//  predextract<bfs, VertexId, SizeT, Value, g_mark_predecessor, g_with_value > (csr_problem, device_id, d_predlist);
	pred_extract(csr_problem.graph_slices[0]->nodes,
			csr_problem.graph_slices[0]->d_column_offsets,
			csr_problem.graph_slices[0]->d_row_indices,
			csr_problem.graph_slices[0]->vertex_list.d_labels, device_id,
			d_predlist);

//  int* test_vid = new int[csr_graph.nodes];
//  hipMemcpy(test_vid, d_predlist, csr_graph.nodes * sizeof (int), hipMemcpyDeviceToHost);
//  printf("d_predlist: ");
//  for (int i = 0; i < (csr_graph.nodes); ++i)
//  {
//    printf("%d, ", test_vid[i]);
//  }
//  printf("\n");
//  delete[] test_vid;

	Value* h_values = (Value*) malloc(sizeof(Value) * csr_graph.nodes);
	csr_problem.ExtractResults(h_values);

	//compute GTEPS
	long long te = 0;
	for (int i = 0; i < csr_graph.nodes; i++) {
		if (h_values[i] != -1) {
			te += csr_graph.row_offsets[i + 1] - csr_graph.row_offsets[i];
		}
	}

	printf("Traversed edge: %lld\n", te);

	if (strcmp(source_file_name, "") == 0 && run_CPU) {
		const bool ok = correctTest(csr_graph.nodes, reference_labels,
				h_values);
		free(reference_labels);
		if (!ok) {
			fprintf(stderr, "correctness test failed.");
			exit(1);
		}
	}

	if (outFileName) {
		FILE* f = fopen(outFileName, "w");
		for (int i = 0; i < csr_graph.nodes; ++i) {
			fprintf(f, "%d\n", h_values[i]);
		}

		fclose(f);
	}

	return 0;
}
